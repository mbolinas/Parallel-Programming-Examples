#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include "book.h"

int n, m, i, j;

__global__ void l2Norm(double * m_d, int numRow, int numCol){  
  int tid = blockIdx.x;//tid = row #
  double partialsum = 0;
  for(int i = 0; i < numCol; i++){
    partialsum = partialsum + (m_d[(tid * numCol) + i] * m_d[(tid * numCol) + i]);
  }
  m_d[(tid * numCol)] = sqrt(partialsum);
}


int main(int argc, char * argv[]) {

  assert(argc==3);


  n = atoi(argv[1]);//rows
  m = atoi(argv[2]);//columns


  double *d_a;

  double *arr1 = (double *) malloc((n * m) * sizeof(*arr1));

  
  printf("test before main loops \n");
  fflush(stdout);


  //"randomly" allocate the array
  int count = 0;
  for(int i = 0; i < n * m; i++){
    arr1[i] = (i % m) + (count * ((n + 1) / m));
    //printf("%f ", arr1[i]);
    
    if(i % m == m - 1){
      //printf("\n");
    }
    
    count = (count + 1) % (m - 1);
  }

  printf("\n");

  clock_t tic = clock();

  HANDLE_ERROR(hipMalloc( (void**) &d_a, (n * m) * sizeof(double)));
  HANDLE_ERROR(hipMemcpy( d_a, arr1, (n * m) * sizeof(double), hipMemcpyHostToDevice));

  l2Norm<<<n, 1>>>(d_a, n, m);

  HANDLE_ERROR(hipMemcpy( arr1, d_a, (n * m) * sizeof(double), hipMemcpyDeviceToHost));

  double total = 0;
  for(int i = 0; i < n * m; i = i + m){
    total = total + arr1[i];
  }
    
  clock_t toc = clock();
  double time_elapsed = (double) (toc - tic) / CLOCKS_PER_SEC;

  printf("Total: %lf (%f sec) \n", total, time_elapsed);
  fflush(stdout);

  hipFree(d_a);
  return 0;
}
